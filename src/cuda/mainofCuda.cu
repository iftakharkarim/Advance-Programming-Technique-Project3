#include "hip/hip_runtime.h"
#include "input_image.h"
#include "input_image.cu"
#include "complex.h"
#include "complex.cu"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cmath>
using namespace std;

#define M_PI 3.14159265358979323846

// device function which does row wise DFT
__global__ doDFT1(Complex *gpuArray1, Complex *gpuArray2, int width, int height, float M_PI) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	index = min(index, width*height - 1);
	int y = index / width; // current row
	Complex value = new Complex();
	for(int k = 0; k < width; k++) {
		Complex w = exp(new Complex(0, -2*M_PI*k*y/(width))); // twiddler factor
		value = value + gpuArray2[y*width + k] * w;
	}
	gpuArray1[index] = value;
}
// device function which does column wise DFT
__global__ doDFT2(Complex *gpuArray1, Complex *gpuArray2, int width, int height, float M_PI) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	index = min(index, width*height - 1);
	int x = index % width; // current column
	Complex value = new Complex();
	for(int k = 0; k < height; k++) {
		Complex w = exp(new Complex(0, -2*M_PI*k*x/(height))); // twiddler factor
		value = value + gpuArray2[x + width*k] * w;
	}
	gpuArray1[index] = value;
}


int main(int argc, char const **argv)
{	
	if(argc == 4){
		ifstream inFile(argv[2]);
		ifstream outFile(argv[3]);
	}else{
		cout<<"Expected ./file forward/reverse inputfile outputfile"<<endl;
	}
	
	InputImage input = InputImage(argv[2]);
	int width = input.get_width();
	int height = input.get_height();
	cout<<"w  "<<width<<"  h  "<<height<<endl;


	cout<<input.get_image_data()<<endl;
	Complex *inputArray = input.get_image_data();
	cout<<*inputArray<<endl;
	cout<<*(inputArray+1)<<endl;

	// define, allocate and copy data for gpu
	Complex *gpuArray1, gpuArray2;
	int size = w*h*sizeof(Complex);
	hipMalloc(&gpuArray1, size);
	hipMalloc(&gpuArray2, size);

	hipMemcpy(gpuArray1, &inputArray, size, hipMemcpyHostToDevice);
	hipMemcpy(gpuArray2, &inputArray, size, hipMemcpyHostToDevice);

	// do the first dft which is row wise
	int numOfThreads = (width*height < 1024)? 512 : 1024;
	doDFT1<<<(size/numOfThreads) + 1 , numOfThreads >>>(gpuArray1, gpuArray2, width, height, M_PI);
	hipMemcpy(inputArray, gpuArray1, size, hipMemcpyDeviceToHost);

	// copy everything back to gpu in the second array
	hipMemcpy(gpuArray2, &inputArray, size, hipMemcpyHostToDevice);

	// do the second dft which is column wise
	doDFT2<<<(size/numOfThreads)+1 , numOfThreads>>>(gpuArray1, gpuArray2, width, height, M_PI);

	// copy the final result
	hipMemcpy(inputArray, gpuArray1, size, hipMemcpyDeviceToHost);
	
	// free all the memory
	hipFree(gpuArray1), hipFree(gpuArray2);

	// write output into the given text file
	input.save_image_data(argv[2], input, width, height);
	return 0;
}